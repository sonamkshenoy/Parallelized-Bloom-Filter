#include "hip/hip_runtime.h"
#include "bloomfilter.h"
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <bitset>
#include <cstring>
#include <ctime>
#include <omp.h>
#include <inttypes.h>
#include <iomanip>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <cstdio>

using namespace std;

#define BIG_CONSTANT(x) (x)
#define ROTL64(x,y) rotl64(x,y)
#define ROTL642(x,y) rotl642(x,y)

#define	FORCE_INLINE inline __attribute__((always_inline))

#define BIT_ARRAY_SIZE  1000
#define SEED_VALUE_1 27
#define SEED_VALUE_2 58
#define SEED_VALUE_3 99

const int MAX = 26;



__device__ inline uint64_t rotl64(uint64_t x, int8_t r){
  return (x << r) | (x >> (64 - r));
}

inline uint64_t rotl642(uint64_t x, int8_t r){
  return (x << r) | (x >> (64 - r));
}

__device__ FORCE_INLINE uint64_t fmix64 ( uint64_t k )
{
  k ^= k >> 33;
  k *= BIG_CONSTANT(0xff51afd7ed558ccd);
  k ^= k >> 33;
  k *= BIG_CONSTANT(0xc4ceb9fe1a85ec53);
  k ^= k >> 33;

  return k;
}

FORCE_INLINE uint64_t fmix642 ( uint64_t k )
{
  k ^= k >> 33;
  k *= BIG_CONSTANT(0xff51afd7ed558ccd);
  k ^= k >> 33;
  k *= BIG_CONSTANT(0xc4ceb9fe1a85ec53);
  k ^= k >> 33;

  return k;
}

__device__ FORCE_INLINE uint64_t getblock64 ( const uint64_t * p, int i )
{
  return p[i];
}


FORCE_INLINE uint64_t getblock642 ( const uint64_t * p, int i )
{
  return p[i];
}

__device__ void MurmurHash3_x64_128(const void* key, const int len, const uint32_t seed, uint64_t* hash){

  
  const uint8_t* data = (const uint8_t*)key;
  
  const int nblocks = len/16;

  uint64_t h1 = seed;
  uint64_t h2 = seed;

  const uint64_t c1 = BIG_CONSTANT(0x87c37b91114253d5);
  const uint64_t c2 = BIG_CONSTANT(0x4cf5ad432745937f);

  //------------
  // body

  const uint64_t *blocks = (const uint64_t *)(data);

  
  for(int i = 0; i < nblocks; i++){


    uint64_t k1 = getblock64(blocks,i*2+0);
    uint64_t k2 = getblock64(blocks,i*2+1);

    k1 *= c1;
    k1  = ROTL64(k1,31);
    k1 *= c2;
    h1 ^= k1;

    h1 = ROTL64(h1,27);
    h1 += h2;
    h1 = h1*5+0x52dce729;

    k2 *= c2;
    k2  = ROTL64(k2,33);
    k2 *= c1;
    h2 ^= k2;

    h2 = ROTL64(h2,31);
    h2 += h1;
    h2 = h2*5+0x38495ab5;
  }
  

  //----------
  // tail

  const uint8_t * tail = (const uint8_t*)(data + nblocks*16);
  

  uint64_t k1 = 0;
  uint64_t k2 = 0;

  switch(len & 15){
    case 15: k2 ^= ((uint64_t)tail[14]) << 48;
    case 14: k2 ^= ((uint64_t)tail[13]) << 40;
    case 13: k2 ^= ((uint64_t)tail[12]) << 32;
    case 12: k2 ^= ((uint64_t)tail[11]) << 24;
    case 11: k2 ^= ((uint64_t)tail[10]) << 16;
    case 10: k2 ^= ((uint64_t)tail[ 9]) << 8;
    case  9: k2 ^= ((uint64_t)tail[ 8]) << 0;
             k2 *= c2; k2  = ROTL64(k2,33); k2 *= c1; h2 ^= k2;

    case  8: k1 ^= ((uint64_t)tail[ 7]) << 56;
    case  7: k1 ^= ((uint64_t)tail[ 6]) << 48;
    case  6: k1 ^= ((uint64_t)tail[ 5]) << 40;
    case  5: k1 ^= ((uint64_t)tail[ 4]) << 32;
    case  4: k1 ^= ((uint64_t)tail[ 3]) << 24;
    case  3: k1 ^= ((uint64_t)tail[ 2]) << 16;
    case  2: k1 ^= ((uint64_t)tail[ 1]) << 8;
    case  1: k1 ^= ((uint64_t)tail[ 0]) << 0;
             k1 *= c1; k1  = ROTL64(k1,31); k1 *= c2; h1 ^= k1;
  };

  //----------
  // finalization

  h1 ^= len; h2 ^= len;

  h1 += h2;
  h2 += h1;

  h1 = fmix64(h1);
  h2 = fmix64(h2);

  h1 += h2;
  h2 += h1;

  ((uint64_t*)hash)[0] = h1;
  ((uint64_t*)hash)[1] = h2;


}

void MurmurHash3_x64_1282(const void* key, const int len, const uint32_t seed, uint64_t* hash){

  
  const uint8_t* data = (const uint8_t*)key;
  
  const int nblocks = len/16;

  uint64_t h1 = seed;
  uint64_t h2 = seed;

  const uint64_t c1 = BIG_CONSTANT(0x87c37b91114253d5);
  const uint64_t c2 = BIG_CONSTANT(0x4cf5ad432745937f);

  //------------
  // body

  const uint64_t *blocks = (const uint64_t *)(data);

  
  for(int i = 0; i < nblocks; i++){


    uint64_t k1 = getblock642(blocks,i*2+0);
    uint64_t k2 = getblock642(blocks,i*2+1);

    k1 *= c1;
    k1  = ROTL642(k1,31);
    k1 *= c2;
    h1 ^= k1;

    h1 = ROTL642(h1,27);
    h1 += h2;
    h1 = h1*5+0x52dce729;

    k2 *= c2;
    k2  = ROTL642(k2,33);
    k2 *= c1;
    h2 ^= k2;

    h2 = ROTL642(h2,31);
    h2 += h1;
    h2 = h2*5+0x38495ab5;
  }
  

  //----------
  // tail

  const uint8_t * tail = (const uint8_t*)(data + nblocks*16);
  

  uint64_t k1 = 0;
  uint64_t k2 = 0;

  switch(len & 15){
    case 15: k2 ^= ((uint64_t)tail[14]) << 48;
    case 14: k2 ^= ((uint64_t)tail[13]) << 40;
    case 13: k2 ^= ((uint64_t)tail[12]) << 32;
    case 12: k2 ^= ((uint64_t)tail[11]) << 24;
    case 11: k2 ^= ((uint64_t)tail[10]) << 16;
    case 10: k2 ^= ((uint64_t)tail[ 9]) << 8;
    case  9: k2 ^= ((uint64_t)tail[ 8]) << 0;
             k2 *= c2; k2  = ROTL642(k2,33); k2 *= c1; h2 ^= k2;

    case  8: k1 ^= ((uint64_t)tail[ 7]) << 56;
    case  7: k1 ^= ((uint64_t)tail[ 6]) << 48;
    case  6: k1 ^= ((uint64_t)tail[ 5]) << 40;
    case  5: k1 ^= ((uint64_t)tail[ 4]) << 32;
    case  4: k1 ^= ((uint64_t)tail[ 3]) << 24;
    case  3: k1 ^= ((uint64_t)tail[ 2]) << 16;
    case  2: k1 ^= ((uint64_t)tail[ 1]) << 8;
    case  1: k1 ^= ((uint64_t)tail[ 0]) << 0;
             k1 *= c1; k1  = ROTL642(k1,31); k1 *= c2; h1 ^= k1;
  };

  //----------
  // finalization

  h1 ^= len; h2 ^= len;

  h1 += h2;
  h2 += h1;

  h1 = fmix642(h1);
  h2 = fmix642(h2);

  h1 += h2;
  h2 += h1;

  ((uint64_t*)hash)[0] = h1;
  ((uint64_t*)hash)[1] = h2;


}

string genRandomString(int n) 
{ 
    char alphabet[MAX] = { 'a', 'b', 'c', 'd', 'e', 'f', 'g', 
                          'h', 'i', 'j', 'k', 'l', 'm', 'n',  
                          'o', 'p', 'q', 'r', 's', 't', 'u', 
                          'v', 'w', 'x', 'y', 'z' }; 
  
    string res = ""; 
    for (int i = 0; i < n; i++)  
        res = res + alphabet[rand() % MAX]; 
      
    return res; 
}

__global__ void insertInHashTable(int* HashTable, char* key, int length){
  
  // Calculate 3 hashes and insert
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if(idx == 0){
  uint64_t hash1[2];
  MurmurHash3_x64_128(key, length, SEED_VALUE_1, hash1);
  int bit1 = (hash1[0] % BIT_ARRAY_SIZE + hash1[1] % BIT_ARRAY_SIZE) % BIT_ARRAY_SIZE;
  printf("%d", bit1);
  HashTable[bit1] = 1;
  
  }

  if(idx == 1){
  uint64_t hash2[2];
  MurmurHash3_x64_128(key, length, SEED_VALUE_2, hash2);
  int bit2 = (hash2[0] % BIT_ARRAY_SIZE + hash2[1] % BIT_ARRAY_SIZE) % BIT_ARRAY_SIZE;
  HashTable[bit2] = 1;
  //printf("bit 2: %d", bit2);
  }

  if(idx == 2){
  uint64_t hash3[2];
  MurmurHash3_x64_128(key, length, SEED_VALUE_3, hash3);
  int bit3 = (hash3[0] % BIT_ARRAY_SIZE + hash3[1] % BIT_ARRAY_SIZE) % BIT_ARRAY_SIZE;
  HashTable[bit3] = 1;
  //printf("bit 3: %d", bit3);
  }
}


void checkIfPresent(int* HashTable, char* key, int length){
  
//   // Calculate 3 hashes and check bit
//   int idx = threadIdx.x + blockIdx.x * blockDim.x;


//   if(idx == 0){
//   printf("Entering2\n");
  uint64_t hash1[2];
  MurmurHash3_x64_1282(key, length, SEED_VALUE_1, hash1);
  int bit1 = (hash1[0] % BIT_ARRAY_SIZE + hash1[1] % BIT_ARRAY_SIZE) % BIT_ARRAY_SIZE;
//   printf("bit 1: %d", bit1);
// }

//   if(idx == 1){
  uint64_t hash2[2];
  MurmurHash3_x64_1282(key, length, SEED_VALUE_2, hash2);
  int bit2 = (hash2[0] % BIT_ARRAY_SIZE + hash2[1] % BIT_ARRAY_SIZE) % BIT_ARRAY_SIZE;
//   printf("bit 2: %d", bit2);
// }

//   if(idx == 2){
  uint64_t hash3[2];
  MurmurHash3_x64_1282(key, length, SEED_VALUE_3, hash3);
  int bit3 = (hash3[0] % BIT_ARRAY_SIZE + hash3[1] % BIT_ARRAY_SIZE) % BIT_ARRAY_SIZE;
//   printf("bit 3: %d", bit3);
//   }

//   // __syncthreads();
  if(HashTable[bit1] == 1 && HashTable[bit2] == 1 && HashTable[bit3] == 1){
    printf("%s might be present\n", key);
  }
  else{
    printf("%s is definitely not present\n", key);
  }
}

int main(){

  time_t start, end;

  time(&start);

  int* HashTable = (int*)calloc(BIT_ARRAY_SIZE, sizeof(int));

  int* d_HashTable;
  hipMalloc((void**)&d_HashTable, BIT_ARRAY_SIZE*sizeof(int));
  hipMemset(d_HashTable, 0, BIT_ARRAY_SIZE*sizeof(int));
  cout << d_HashTable[0];
  //  for(int i=0; i<BIT_ARRAY_SIZE; i++){
  //   cout << d_HashTable[i];
  // }

  // copy to DEVICE
  hipMemcpy(d_HashTable, HashTable, BIT_ARRAY_SIZE*sizeof(int), hipMemcpyHostToDevice);

  cout << sizeof(d_HashTable) << endl;

  string str = "thefirsthashfunctionjdfsjldkjklsjlfjdslkjflsjwjadjsaijadijasjdoiajsdoiannewfjsoinsfnoiesfinosen";
  int len = str.length();
  cout << len << endl;
  char *cstr = new char[str.length() + 1];
  strcpy(cstr, str.c_str());
  
  
  insertInHashTable<<<1,3>>>(d_HashTable, cstr, len);
  hipDeviceSynchronize();

  hipMemcpy(HashTable, d_HashTable, BIT_ARRAY_SIZE*sizeof(int), hipMemcpyDeviceToHost);
 
  cout << endl;
  checkIfPresent(HashTable, cstr, len);
  // /hipDeviceSynchronize();
  hipFree(d_HashTable);
  

  time(&end);
  double timeTaken = double(end - start);
  //cout << "Time taken for inserting " << numIterations <<  " records in unparallelized version: " << fixed << timeTaken << setprecision(9);
  //cout << "s" << endl;

}