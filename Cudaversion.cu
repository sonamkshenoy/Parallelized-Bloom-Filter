#include "hip/hip_runtime.h"
#include "bloomfilter.h"
#include <stdlib.h>
#include <iostream>
#include <semaphore.h>
#include <vector>
#include <bitset>
#include <cstring>
#include <ctime>
#include <omp.h>
#include <inttypes.h>
#include <iomanip>
#include <iomanip>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <cstdio>
#include <chrono>
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <string>
#include <hip/hip_runtime_api.h>

using namespace std;

#define BIG_CONSTANT(x) (x)
#define ROTL64(x,y) rotl64(x,y)
#define	FORCE_INLINE inline __attribute__((always_inline))

#define BIT_ARRAY_SIZE  100000
#define SEED_VALUE_1 27
#define SEED_VALUE_2 58
#define SEED_VALUE_3 99


const int MAX = 26;
sem_t semaphore;

__device__ inline uint64_t rotl64(uint64_t x, int8_t r){
  return (x << r) | (x >> (64 - r));
}

__device__ FORCE_INLINE uint64_t fmix64 ( uint64_t k )
{
  k ^= k >> 33;
  k *= BIG_CONSTANT(0xff51afd7ed558ccd);
  k ^= k >> 33;
  k *= BIG_CONSTANT(0xc4ceb9fe1a85ec53);
  k ^= k >> 33;

  return k;
}


__device__ FORCE_INLINE uint64_t getblock64 ( const uint64_t * p, int i )
{
  return p[i];
}

__device__ void MurmurHash3_x64_128(const void* key, const int len, const uint32_t seed, uint64_t* hash, uint64_t* kvalues){

  const uint8_t* data = (const uint8_t*)key;
  const int nblocks = len/16;
  uint64_t h1 = seed;
  uint64_t h2 = seed;
  uint64_t c1;
  uint64_t c2;
  c1 = BIG_CONSTANT(0x87c37b91114253d5);
  c2 = BIG_CONSTANT(0x4cf5ad432745937f);
  //const uint64_t *blocks = (const uint64_t *)(data);

  uint64_t k1, k2;

  for(int i = 0; i < nblocks; i++){
    k1 = kvalues[i*2 + 0];
    k2 = kvalues[i*2 + 1];

    h1 ^= k1;

    h1 = ROTL64(h1,27);
    h1 += h2;
    h1 = h1*5+0x52dce729;

    h2 ^= k2;

    h2 = ROTL64(h2,31);
    h2 += h1;
    h2 = h2*5+0x38495ab5;
  }


  //----------
  // tail

  const uint8_t * tail = (const uint8_t*)(data + nblocks*16);

  // uint64_t 
  k1 = 0;
  //uint64_t 
  k2 = 0;

  switch(len & 15){
    case 15: k2 ^= ((uint64_t)tail[14]) << 48;
    case 14: k2 ^= ((uint64_t)tail[13]) << 40;
    case 13: k2 ^= ((uint64_t)tail[12]) << 32;
    case 12: k2 ^= ((uint64_t)tail[11]) << 24;
    case 11: k2 ^= ((uint64_t)tail[10]) << 16;
    case 10: k2 ^= ((uint64_t)tail[ 9]) << 8;
    case  9: k2 ^= ((uint64_t)tail[ 8]) << 0;
             k2 *= c2; k2  = ROTL64(k2,33); k2 *= c1; h2 ^= k2;

    case  8: k1 ^= ((uint64_t)tail[ 7]) << 56;
    case  7: k1 ^= ((uint64_t)tail[ 6]) << 48;
    case  6: k1 ^= ((uint64_t)tail[ 5]) << 40;
    case  5: k1 ^= ((uint64_t)tail[ 4]) << 32;
    case  4: k1 ^= ((uint64_t)tail[ 3]) << 24;
    case  3: k1 ^= ((uint64_t)tail[ 2]) << 16;
    case  2: k1 ^= ((uint64_t)tail[ 1]) << 8;
    case  1: k1 ^= ((uint64_t)tail[ 0]) << 0;
             k1 *= c1; k1  = ROTL64(k1,31); k1 *= c2; h1 ^= k1;
  };

  //----------
  // finalization

  h1 ^= len; h2 ^= len;

  h1 += h2;
  h2 += h1;

  h1 = fmix64(h1);
  h2 = fmix64(h2);

  h1 += h2;
  h2 += h1;

  ((uint64_t*)hash)[0] = h1;
  ((uint64_t*)hash)[1] = h2;

}

string genRandomString(int n) 
{ 
    char alphabet[MAX] = { 'a', 'b', 'c', 'd', 'e', 'f', 'g', 
                          'h', 'i', 'j', 'k', 'l', 'm', 'n',  
                          'o', 'p', 'q', 'r', 's', 't', 'u', 
                          'v', 'w', 'x', 'y', 'z' }; 
  
    string res = ""; 

    for (int i = 0; i < n; i++)  
        res = res + alphabet[rand() % MAX]; 
      
    return res; 
} 

__device__ void insertInHashTable(char* key, int length, int* d_bitArray, int idx){
  
  // Calculate 3 hashes and insert
  uint64_t hash1[2];
  uint64_t hash2[2];
  uint64_t hash3[2];
  int bit1, bit2, bit3;


  const uint8_t* data = (const uint8_t*)key;
  const int nblocks = length/16;

  uint64_t c1;
  uint64_t c2;
  c1 = BIG_CONSTANT(0x87c37b91114253d5);
  c2 = BIG_CONSTANT(0x4cf5ad432745937f);
  const uint64_t *blocks = (const uint64_t *)(data);
  
  uint64_t k1, k2;

  uint64_t kvalues[10];

  for(int i = 0; i < nblocks; i++){
    k1 = getblock64(blocks,i*2+0);
    k1 *= c1;
    k1  = ROTL64(k1,31);
    k1 *= c2;

    k2 = getblock64(blocks,i*2+1);
    k2 *= c2;
    k2  = ROTL64(k2,33);
    k2 *= c1;

    kvalues[i*2 + 0] = k1;
    kvalues[i*2 + 1] = k2;
  }

MurmurHash3_x64_128(key, length, SEED_VALUE_1, hash1, kvalues);
bit1 = (hash1[0] % BIT_ARRAY_SIZE + hash1[1] % BIT_ARRAY_SIZE) % BIT_ARRAY_SIZE;

MurmurHash3_x64_128(key, length, SEED_VALUE_2, hash2, kvalues);
bit2 = (hash2[0] % BIT_ARRAY_SIZE + hash2[1] % BIT_ARRAY_SIZE) % BIT_ARRAY_SIZE;

MurmurHash3_x64_128(key, length, SEED_VALUE_3, hash3, kvalues);
bit3 = (hash3[0] % BIT_ARRAY_SIZE + hash3[1] % BIT_ARRAY_SIZE) % BIT_ARRAY_SIZE;  



  // cout << "Bits set are: " << bit1 << "," << bit2 << " and " << bit3 << "\n";

  // printf("bit array at %d: %d\n", idx*3+0, bit1);
  // printf("bit array at %d: %d\n", idx*3+1, bit2);
  // printf("bit array at %d: %d\n", idx*3+2, bit3);
  // d_HashTable[bit1] = 1;
  // d_HashTable[bit2] = 1;
  // d_HashTable[bit3] = 1;

  
  d_bitArray[idx*3+0] = bit1;
  d_bitArray[idx*3+1] = bit2;
  d_bitArray[idx*3+2] = bit3;


  
  

  //cout << "Set bits: " << bit1 << ", " << bit2 << ", " << bit3 << "\n";
}


/*
void checkIfPresent(bitset<BIT_ARRAY_SIZE> HashTable, char* key, int length){
  
  // Calculate 3 hashes and check bit

  uint64_t hash1[2];
  MurmurHash3_x64_128(key, length, SEED_VALUE_1, hash1);
  int bit1 = (hash1[0] % BIT_ARRAY_SIZE + hash1[1] % BIT_ARRAY_SIZE) % BIT_ARRAY_SIZE;

  uint64_t hash2[2];
  MurmurHash3_x64_128(key, length, SEED_VALUE_2, hash2);
  int bit2 = (hash2[0] % BIT_ARRAY_SIZE + hash2[1] % BIT_ARRAY_SIZE) % BIT_ARRAY_SIZE;

  uint64_t hash3[2];
  MurmurHash3_x64_128(key, length, SEED_VALUE_3, hash3);
  int bit3 = (hash3[0] % BIT_ARRAY_SIZE + hash3[1] % BIT_ARRAY_SIZE) % BIT_ARRAY_SIZE;
  
  if(HashTable.test(bit1) == 1 && HashTable.test(bit2) == 1 && HashTable.test(bit3) == 1){
    cout << key << " might be present" << "\n";
  }
  else{
    cout << key << " is definitely not present" << "\n";
  }
}*/

__device__ char* getword(char* d_wordsToInsert, int idx, int lenOfWord){
  char* temp = new char[lenOfWord + 1];
  for(int i=0; i<lenOfWord; i++){
    temp[i] = d_wordsToInsert[idx*lenOfWord+i];
  }
  temp[lenOfWord] = '\0';
  
  return temp;
}

__global__ void parallelInsertion(char* d_wordsToInsert, int lenOfWord, int numIterations, int* d_bitArray){
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int gridStride = blockDim.x * gridDim.x;

  for(int i=idx; i<numIterations; i += gridStride){
    char* cstr = getword(d_wordsToInsert, i, lenOfWord);
    insertInHashTable(cstr, lenOfWord, d_bitArray, i);
  }
}

int main(int argc, char**argv){

  int lenOfWord = atoi(argv[1]);
  string str;
  int numIterations = atoi(argv[2]);
  int numBlocks = atoi(argv[3]);
  int numThreads = atoi(argv[4]);


  char wordsToInsert[lenOfWord * numIterations];

  for(int i = 0; i < numIterations; i++){
      str = genRandomString(lenOfWord);
      char* cstr = new char[lenOfWord + 1];
      strcpy(cstr, str.c_str());

      for(int j = 0; j < lenOfWord; j++){
          wordsToInsert[i*lenOfWord+j] = cstr[j];
    }
  }
  
  int bitArray[3*numIterations];
  int* d_bitArray;
  hipMalloc((void**)&d_bitArray, 3*numIterations*sizeof(int));
  hipMemcpy(d_bitArray, bitArray, 3*numIterations*sizeof(int), hipMemcpyHostToDevice);

  char* d_wordsToInsert;
  hipMalloc((void**)&d_wordsToInsert, lenOfWord*numIterations*sizeof(char));
  hipMemcpy(d_wordsToInsert, wordsToInsert, lenOfWord*numIterations*sizeof(char), hipMemcpyHostToDevice);

  
  //time and call function here
  auto t_start = std::chrono::high_resolution_clock::now();
 
  parallelInsertion<<<numBlocks, numThreads>>>(d_wordsToInsert, lenOfWord, numIterations, d_bitArray);
  hipDeviceSynchronize();
  
  auto t_end = std::chrono::high_resolution_clock::now();

  

  hipMemcpy(bitArray, d_bitArray, 3*numIterations*sizeof(int), hipMemcpyDeviceToHost);

  
  // hipFree(d_HashTable);
  hipFree(d_wordsToInsert);
  hipFree(d_bitArray);
  hipProfilerStop();
  //hipFree(d_kvalues);

  //free(HashTable);
  
  double elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end-t_start).count();

  // cout << "Time taken for inserting " << numIterations <<  " records in CUDA parallelized version: " << elapsed_time_ms << setprecision(9);
  // cout << " ms" << endl;

  

  std::ofstream outfile;
  outfile.open("./Times/cuda_times.txt", std::ios_base::app);
  outfile << lenOfWord << ":" << numIterations << ":" << numBlocks << ":" << numThreads << ":" << elapsed_time_ms << endl;

  return 0;
}
